#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <sys/time.h>
#include <cblas.h>

//nvcc -o matmul matmul.cu -lcublas -lcuda -lopenblas

#define TIME_START gettimeofday(&t_start, NULL);
#define TIME_END(name)    gettimeofday(&t_end, NULL); \
                    elapsedTime = (t_end.tv_sec - t_start.tv_sec) * 1000.0;   \
                    elapsedTime += (t_end.tv_usec - t_start.tv_usec) / 1000.0;  \
                    printf(#name " Time = %f ms.\n", elapsedTime);

typedef struct 
{
    size_t rows;
    size_t cols;
    float * data; // CPU memory
    float * data_device; // GPU memory
} Matrix;

Matrix * createMatrix(size_t r, size_t c)
{
    size_t len = r * c;
    if(len == 0)
    {
        fprintf(stderr, "Invalid size. The input should be > 0.\n");
        return NULL;
    }
    Matrix * p  = (Matrix *) malloc(sizeof(Matrix));
    if (p == NULL)
    {
        fprintf(stderr, "Allocate host memory failed.\n");
        goto ERR_TAG;
    }
    p->rows = r;
    p->cols = c;
    p->data = (float*)malloc(sizeof(float) * len);
    if(p->data == NULL)
    {
        fprintf(stderr, "Allocate host memory failed.\n");
        goto ERR_TAG;
    }
    if (hipMalloc(&p->data_device, sizeof(float) * len) != hipSuccess)
    {
        fprintf(stderr, "Allocate device memory failed.\n");
        goto ERR_TAG;
    }
    return p;
ERR_TAG:
    if(p && p->data) free(p->data);
    if(p) free(p);
    return NULL;
}

void freeMatrix(Matrix ** pp)
{
    if(pp == NULL) return;
    Matrix * p = *pp;
    if(p != NULL)
    {
        if(p->data) free(p->data);
        if(p->data_device) hipFree(p->data_device);
    }
    *pp = NULL;
}

// A simple function to set all elements to the same value
bool setMatrix(Matrix * pMat, float val)
{
    if(pMat == NULL)
    {
        fprintf(stderr, "NULL pointer.\n");
        return false;
    }
    size_t len = pMat->rows * pMat->cols;
    for(size_t i = 0; i < len; i++)
        pMat->data[i] = val;

    return true;
}

void matmulCPU(const Matrix * A, const Matrix * B, Matrix * C)
{
    const float alpha = 1.0f;
    const float beta = 0.0f;
    
    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans,
                A->rows, B->cols, A->cols,
                alpha, A->data, A->cols,
                B->data, B->cols,
                beta, C->data, C->cols);
}

// void copyToGPU(Matrix * A, Matrix * B)
// {
//     cudaMemcpy(A->data_device, A->data, A->rows * A->cols * sizeof(float), cudaMemcpyHostToDevice);
//     cudaMemcpy(B->data_device, B->data, B->rows * B->cols * sizeof(float), cudaMemcpyHostToDevice);
// }

// void copyFromGPU(Matrix * C)
// {
//     cudaMemcpy(C->data, C->data_device, C->rows * C->cols * sizeof(float), cudaMemcpyDeviceToHost);
// }

// void matmulGPU(const Matrix * A, const Matrix * B, Matrix * C, cublasHandle_t handle)
// {
//     const float alpha = 1.0f;
//     const float beta = 0.0f;

//     // Perform matrix multiplication: C = alpha * A * B + beta * C
//     cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N,
//                 A->rows, B->cols, A->cols,
//                 &alpha,
//                 A->data_device, A->rows,
//                 B->data_device, B->rows,
//                 &beta,
//                 C->data_device, C->rows);
// }

void matmulGPU(const Matrix * A, const Matrix * B, Matrix * C)
{
    struct timeval t_start, t_end;
    double elapsedTime = 0;

    TIME_START
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    TIME_END(createhandle)

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Copy data to GPU
    TIME_START
    hipMemcpyAsync(A->data_device, A->data, A->rows * A->cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(B->data_device, B->data, B->rows * B->cols * sizeof(float), hipMemcpyHostToDevice);
    TIME_END(CopytoGPU)
    // Start measuring time after data is copied

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    TIME_START
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                A->rows, B->cols, A->cols,
                &alpha,
                A->data_device, A->rows,
                B->data_device, B->rows,
                &beta,
                C->data_device, C->rows);
    TIME_END(RealMultiplication)

    TIME_START
    hipDeviceSynchronize(); // Ensure the kernel has completed
    TIME_END(Synchronize)
    //TIME_END(matmulGPU)

    // Copy result back to CPU
    TIME_START
    hipMemcpyAsync(C->data, C->data_device, C->rows * C->cols * sizeof(float), hipMemcpyDeviceToHost);
    TIME_END(CopyfromGPU)

    TIME_START
    hipblasDestroy(handle);
    TIME_END(Destroyhandle)
}

int main()
{
    struct timeval t_start, t_end;
    double elapsedTime = 0;

    int dev_count = 0;
    int dev_id = 0;
    hipGetDeviceCount(&dev_count);
    hipSetDevice(0);
    hipGetDevice(&dev_id);
    printf("You have %d cuda devices.\n", dev_count);
    printf("You are using device %d.\n", dev_id);

    size_t size = 4096;
    Matrix * A = createMatrix(size, size);
    Matrix * B = createMatrix(size, size);
    Matrix * C = createMatrix(size, size);

    setMatrix(A, 1.0f);
    setMatrix(B, 2.0f);

    // Measure CPU matrix multiplication time
    TIME_START
    matmulCPU(A, B, C);
    TIME_END(matmulCPU)
    printf("  Result (CPU) = [%.1f, ..., %.1f]\n", C->data[0], C->data[C->rows * C->cols - 1]);

    // Copy data to GPU
    TIME_START
    //copyToGPU(A, B);
    //TIME_END(copyToGPU)

    //TIME_START
    //cublasHandle_t handle;
    //cublasCreate(&handle);
    //TIME_END(createhandle)

    // Measure GPU matrix multiplication time
   // TIME_START
    matmulGPU(A, B, C);
    //cudaDeviceSynchronize(); // Ensure the kernel has completed
    //TIME_END(matmulGPU)

    // Copy result back to CPU
    //TIME_START
    //copyFromGPU(C);
    TIME_END(matmulGPU)
    printf("  Result (GPU) = [%.1f, ..., %.1f]\n", C->data[0], C->data[C->rows * C->cols - 1]);

    //TIME_START
    //cublasDestroy(handle);
    //TIME_END(destroy)

    freeMatrix(&A);
    freeMatrix(&B);
    freeMatrix(&C);
    return 0;
}
