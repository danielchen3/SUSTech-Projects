#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h> 

#define TIME_START gettimeofday(&t_start, NULL);
#define TIME_END(name)    gettimeofday(&t_end, NULL); \
                    elapsedTime = (t_end.tv_sec - t_start.tv_sec) * 1000.0;   \
                    elapsedTime += (t_end.tv_usec - t_start.tv_usec) / 1000.0;  \
                    printf(#name " Time = %f ms.\n", elapsedTime);

typedef struct 
{
    size_t rows;
    size_t cols;
    float * data; // CPU memory
    float * data_device; // GPU memory
} Matrix;

Matrix * createMatrix(size_t r, size_t c)
{
    size_t len = r * c;
    if(len == 0)
    {
        fprintf(stderr, "Invalid size. The input should be > 0.\n");
        return NULL;
    }
    Matrix * p  = (Matrix *) malloc(sizeof(Matrix));
    if (p == NULL)
    {
        fprintf(stderr, "Allocate host memory failed.\n");
        goto ERR_TAG;
    }
    p->rows = r;
    p->cols = c;
    p->data = (float*)malloc(sizeof(float)*len);
    if(p->data == NULL)
    {
        fprintf(stderr, "Allocate host memory failed.\n");
        goto ERR_TAG;
    }
    if (hipMalloc (&p->data_device, sizeof(float) * len) != hipSuccess)
    {
        fprintf(stderr, "Allocate device memory failed.\n");
        goto ERR_TAG;
    }
    return p;
ERR_TAG:
    if(p && p->data) free(p->data);
    if(p) free(p);
    return NULL;
}

void freeMatrix(Matrix ** pp)
{
    if(pp == NULL) return;
    Matrix * p = *pp;
    if(p != NULL)
    {
        if(p->data) free(p->data);
        if(p->data_device) hipFree(p->data_device);
    }
    *pp = NULL;
}

// A simple function to set all elements to the same value
bool setMatrix(Matrix * pMat, float val)
{
    if(pMat == NULL)
    {
        fprintf(stderr, "NULL pointer.\n");
        return false;
    }
    size_t len = pMat->rows * pMat->cols;
    for(size_t i = 0; i < len; i++)
        pMat->data[i] = val;

    return true;
}

bool addCPU(const Matrix * pMat1, const Matrix * pMat2, Matrix * pMat3)
{
    if( pMat1 == NULL 
        || pMat2 == NULL
        || pMat3 == NULL)
    {
        fprintf(stderr, "Null pointer.\n");
        return false;
    }
    if (pMat1->rows != pMat2->rows || pMat1->cols != pMat2->cols ||
        pMat2->rows != pMat3->rows || pMat2->cols != pMat3->cols)
    {
        fprintf(stderr, "The 3 matrices are not in the same size.\n");
        return false;
    }
    size_t len = pMat1->rows * pMat1->cols;
    for (int i = 0; i < len; i++)
        pMat3->data[i] = pMat1->data[i] + pMat2->data[i];
    return true;
}

__global__ void addKernel(const float * input1, const float * input2, float * output, size_t len)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < len)
        output[i] = input1[i] + input2[i];
}

bool addGPU(const Matrix * pMat1, const Matrix * pMat2, Matrix * pMat3)
{
    if( pMat1 == NULL 
        || pMat2 == NULL
        || pMat3 == NULL)
    {
        fprintf(stderr, "Null pointer.\n");
        return false;
    }
    if (pMat1->rows != pMat2->rows || pMat1->cols != pMat2->cols ||
        pMat2->rows != pMat3->rows || pMat2->cols != pMat3->cols)
    {
        fprintf(stderr, "The 3 matrices are not in the same size.\n");
        return false;
    }

    hipError_t ecode = hipSuccess;
    size_t len = pMat1->rows * pMat1->cols;

    hipMemcpy(pMat1->data_device, pMat1->data, sizeof(float)*len, hipMemcpyHostToDevice);
    hipMemcpy(pMat2->data_device, pMat2->data, sizeof(float)*len, hipMemcpyHostToDevice);
    addKernel<<<(len+255)/256, 256>>>(pMat1->data_device, pMat2->data_device, pMat3->data_device, len);
    if ((ecode = hipGetLastError()) != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(ecode));
        return false;
    }
    hipMemcpy(pMat3->data, pMat3->data_device, sizeof(float)*len, hipMemcpyDeviceToHost);

    return true;
}

__global__ void scaleAddKernel(const float *input, float *output, float a, float b, size_t len)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < len)
        output[i] = a * input[i] + b;
}

bool scaleAddGPU(const Matrix * pMatA, Matrix * pMatB, float a, float b)
{
    if (pMatA == NULL || pMatB == NULL)
    {
        fprintf(stderr, "Null pointer.\n");
        return false;
    }
    if (pMatA->rows != pMatB->rows || pMatA->cols != pMatB->cols)
    {
        fprintf(stderr, "The matrices are not in the same size.\n");
        return false;
    }

    size_t len = pMatA->rows * pMatA->cols;

    //进行memory的copy，相当于将数据载入到GPU中
    hipMemcpy(pMatA->data_device, pMatA->data, sizeof(float) * len, hipMemcpyHostToDevice);
    scaleAddKernel<<<(len + 255) / 256, 256>>>(pMatA->data_device, pMatB->data_device, a, b, len);

    hipError_t ecode;
    if ((ecode = hipGetLastError()) != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(ecode));
        return false;
    }
    //计算完之后从GPU拿回来
    hipMemcpy(pMatB->data, pMatB->data_device, sizeof(float) * len, hipMemcpyDeviceToHost);

    return true;
}

bool scaleAddCPU(const Matrix * pMatA, Matrix * pMatB, float a, float b)
{
    if (pMatA == NULL || pMatB == NULL)
    {
        fprintf(stderr, "Null pointer.\n");
        return false;
    }
    if (pMatA->rows != pMatB->rows || pMatA->cols != pMatB->cols)
    {
        fprintf(stderr, "The matrices are not in the same size.\n");
        return false;
    }

    size_t len = pMatA->rows * pMatA->cols;
    for (size_t i = 0; i < len; i++)
        pMatB->data[i] = a * pMatA->data[i] + b;

    return true;
}

int main()
{
    struct timeval t_start, t_end;
    double elapsedTime = 0;

    int dev_count = 0;
    int dev_id = 0;
    hipGetDeviceCount(&dev_count);
    hipSetDevice(2);
    hipGetDevice(&dev_id);
    printf("You have %d cuda devices.\n", dev_count);
    printf("You are using device %d.\n", dev_id);

    Matrix * pMatA = createMatrix(4096, 4096);
    Matrix * pMatB = createMatrix(4096, 4096);

    setMatrix(pMatA, 1.1f);

    float a = 2.0f;
    float b = 1.0f;

    TIME_START
    scaleAddCPU(pMatA, pMatB, a, b);
    TIME_END(scaleAddCPU)
    printf("  Result = [%.1f, ..., %.1f]\n", pMatB->data[0], pMatB->data[pMatB->rows*pMatB->cols-1]);

    TIME_START
    scaleAddGPU(pMatA, pMatB, a, b);
    TIME_END(scaleAddGPU)
    printf("  Result = [%.1f, ..., %.1f]\n", pMatB->data[0], pMatB->data[pMatB->rows*pMatB->cols-1]);

    freeMatrix(&pMatA);
    freeMatrix(&pMatB);
    return 0;
}
